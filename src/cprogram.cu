#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h> 
#include <utils.h>
#include <sequential.h>
#include <cuda_prog_11.h>
#include <cuda_prog_m1.h>
#include <cuda_prog_m1_share.h>

int main(int argc, char** argv)
{
    srand(time(NULL));

    int n, k, mode, b;
    parse_arguments(argc, argv, &n, &k, &mode, &b);
    
    int* arr = initiallize_model(n);
    //the second copy is used for validation
    int* copy_arr = (int*) malloc(sizeof(int)*n*n);
    memcpy(copy_arr, arr, n*n*sizeof(int));
    //print_model(n, arr);
    struct timeval t0, t1;
    int validation = 1;
    double elapsed, process;
    switch(mode)
    {
        case 0:
            gettimeofday(&t0, 0);
            arr = sequential_eval(n, k, arr);
            gettimeofday(&t1, 0);
            copy_arr = sequential_eval_ver(n, k, copy_arr);
            validation = compare_matrices(arr, copy_arr, n);
            break;
        case 1:
            gettimeofday(&t0, 0);
            cuda_implementation_v1(arr, n, k, &process);
            gettimeofday(&t1, 0);
            copy_arr = sequential_eval_ver(n, k, copy_arr);
            validation = compare_matrices(arr, copy_arr, n);
            break;
        case 2:
            gettimeofday(&t0, 0);
            cuda_implementation_v2(arr, n, k, b, &process);
            gettimeofday(&t1, 0);
            copy_arr = sequential_eval_ver(n, k, copy_arr);
            validation = compare_matrices(arr, copy_arr, n);
            break;
        case 3:
            gettimeofday(&t0, 0);
            cuda_implementation_v3(arr, n, k, b, &process);
            gettimeofday(&t1, 0);
            copy_arr = sequential_eval_ver(n, k, copy_arr);
            validation = compare_matrices(arr, copy_arr, n);
            break;
    }
    elapsed = ((t1.tv_sec-t0.tv_sec)*1000000 + t1.tv_usec-t0.tv_usec)/1000.0;
    
    //print_model(n, arr);
    
    if(validation == 0)
    {
        save_res(mode, n, k, b, process, elapsed);
        printf("Model evaluated successfully in %.3lfms (actual process %.3lfms)\n", elapsed, process);
    }
    else
        printf("ERROR! Model evaluation failed!\n");
    free(arr);
    free(copy_arr);
    return 0;
}